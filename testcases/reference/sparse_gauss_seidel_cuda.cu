#include <algorithm>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iomanip>
#include <iostream>
#include <numeric>
#include <random>
#include <string>
#include <vector>

// 错误检查宏
#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at %s line %d with error: %s (%d)\n", __FILE__,  \
             __LINE__, hipGetErrorString(status), status);                    \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("cuSPARSE API failed at %s line %d with error: %s (%d)\n",        \
             __FILE__, __LINE__, hipsparseGetErrorString(status), status);      \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

// 类型定义
using data_t = float;

// 常量定义
constexpr data_t EPS = 1e-3f;    // 收敛阈值
constexpr int MAX_ITER = 100000; // 最大迭代次数

// cuSPARSE CSR矩阵包装器
struct CUSPARSEMatrix {
  int rows;
  int cols;
  int nnz;                    // 非零元素数量
  std::vector<data_t> values; // 主机端数据
  std::vector<int> row_offsets;
  std::vector<int> col_indices;

  // 设备端数据
  data_t *d_values = nullptr;
  int *d_row_offsets = nullptr;
  int *d_col_indices = nullptr;

  // cuSPARSE描述符
  hipsparseSpMatDescr_t mat_descr = nullptr;

  CUSPARSEMatrix(int r, int c) : rows(r), cols(c), nnz(0) {
    row_offsets.resize(rows + 1, 0);
  }

  ~CUSPARSEMatrix() {
    if (d_values)
      hipFree(d_values);
    if (d_row_offsets)
      hipFree(d_row_offsets);
    if (d_col_indices)
      hipFree(d_col_indices);
    if (mat_descr)
      hipsparseDestroySpMat(mat_descr);
  }

  void convertFromDense(const std::vector<data_t> &dense_matrix) {
    values.clear();
    col_indices.clear();
    row_offsets[0] = 0;

    for (int i = 0; i < rows; ++i) {
      bool has_diagonal = false;
      for (int j = 0; j < cols; ++j) {
        data_t val = dense_matrix[i * cols + j];
        if (std::abs(val) > EPS) {
          values.push_back(val);
          col_indices.push_back(j);
          if (i == j) {
            has_diagonal = true;
          }
        }
      }
      row_offsets[i + 1] = values.size();

      if (!has_diagonal) {
        std::cerr << "Error: Row " << i << " missing diagonal element!"
                  << std::endl;
        exit(1);
      }
    }
    nnz = values.size();

    std::cout << "Matrix info: " << rows << "x" << cols << ", NNZ: " << nnz
              << std::endl;
    std::cout << "Sparsity: " << (float)nnz / (rows * cols) * 100 << "%"
              << std::endl;
  }

  void moveToDevice() {
    CHECK_CUDA(hipMalloc(&d_values, nnz * sizeof(data_t)));
    CHECK_CUDA(hipMalloc(&d_row_offsets, (rows + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_col_indices, nnz * sizeof(int)));

    CHECK_CUDA(hipMemcpy(d_values, values.data(), nnz * sizeof(data_t),
                          hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_row_offsets, row_offsets.data(),
                          (rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_col_indices, col_indices.data(), nnz * sizeof(int),
                          hipMemcpyHostToDevice));

    CHECK_CUSPARSE(hipsparseCreateCsr(&mat_descr, rows, cols, nnz, d_row_offsets,
                                     d_col_indices, d_values,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
  }

  void multiplyVector(const std::vector<data_t> &x, std::vector<data_t> &b) {
    for (int i = 0; i < rows; ++i) {
      b[i] = 0.0f;
      for (int j = row_offsets[i]; j < row_offsets[i + 1]; ++j) {
        b[i] += values[j] * x[col_indices[j]];
      }
    }
  }
};

// 生成随机稀疏矩阵
std::vector<data_t> generateRandomSparseMatrix(int N) {
  std::vector<data_t> matrix(N * N, 0.0f);
  std::mt19937 gen(13000);
  std::uniform_real_distribution<data_t> dis(-1.0f, 1.0f);

  // 设置目标非零元素数量（大约5%的稀疏度）
  int target_nnz = N * N / 20;
  int current_nnz = 0;

  // 先确保对角元素非零
  for (int i = 0; i < N; ++i) {
    matrix[i * N + i] = 10.0f + std::abs(dis(gen));
    current_nnz++;
  }

  // 随机填充剩余的非零元素
  while (current_nnz < target_nnz) {
    int i = gen() % N;
    int j = gen() % N;
    if (i != j && matrix[i * N + j] == 0.0f) {
      float val = dis(gen);
      matrix[i * N + j] = val;
      current_nnz++;
    }
  }

  // 确保对角占优
  for (int i = 0; i < N; ++i) {
    float row_sum = 0.0f;
    for (int j = 0; j < N; ++j) {
      if (i != j) {
        row_sum += std::abs(matrix[i * N + j]);
      }
    }
    if (row_sum >= std::abs(matrix[i * N + i])) {
      matrix[i * N + i] = row_sum + 1.0f;
    }
  }

  return matrix;
}

// 生成随机精确解向量
std::vector<data_t> generateExactSolution(int N) {
  std::vector<data_t> x(N);
  std::mt19937 gen(13000);
  std::uniform_real_distribution<data_t> dis(-4.0f, 4.0f);
  for (auto &val : x) {
    val = dis(gen);
  }
  return x;
}

// CUDA核函数
__global__ void gaussSeidelKernel(const int N, const data_t *A_values,
                                  const int *A_row_offsets,
                                  const int *A_col_indices, const data_t *b,
                                  data_t *x) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row >= N)
    return;

  data_t sum = 0.0f;
  data_t diag = 0.0f;

  // 找到对角元素位置
  int diag_pos = -1;
  for (int j = A_row_offsets[row]; j < A_row_offsets[row + 1]; j++) {
    if (A_col_indices[j] == row) {
      diag_pos = j;
      diag = A_values[j];
      break;
    }
  }

  if (diag_pos != -1) {
    // 计算非对角元素的和
    for (int j = A_row_offsets[row]; j < A_row_offsets[row + 1]; j++) {
      if (j != diag_pos) {
        int col = A_col_indices[j];
        sum += A_values[j] * x[col];
      }
    }
    // 更新x[row]
    x[row] = (b[row] - sum) / diag;
  }
}

__global__ void computeResidual(int N, const data_t *A_values,
                                const int *A_row_offsets,
                                const int *A_col_indices, const data_t *x,
                                const data_t *b, data_t *r) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row >= N)
    return;

  data_t sum = 0.0f;
  for (int j = A_row_offsets[row]; j < A_row_offsets[row + 1]; j++) {
    sum += A_values[j] * x[A_col_indices[j]];
  }
  r[row] = sum - b[row];
}

__global__ void computeResidualNorm(const data_t *r, data_t *norm, int size) {
  extern __shared__ data_t sdata[];
  int tid = threadIdx.x;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  sdata[tid] = 0.0f;
  if (i < size) {
    sdata[tid] = r[i] * r[i];
  }

  __syncthreads();

  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  if (tid == 0) {
    atomicAdd(norm, sdata[0]);
  }
}

int main(int argc, char **argv) {
  // 参数检查
  if (argc != 2) {
    std::cerr << "Usage: " << argv[0] << " <matrix_dimension>" << std::endl;
    return 1;
  }

  // 解析矩阵维度参数
  int N = std::stoi(argv[1]);
  if (N <= 0) {
    std::cerr << "Matrix dimension must be positive!" << std::endl;
    return 1;
  }

  hipsparseHandle_t cusparse_handle;
  CHECK_CUSPARSE(hipsparseCreate(&cusparse_handle));

  // 生成随机稀疏矩阵
  std::vector<data_t> dense_matrix = generateRandomSparseMatrix(N);

  // 生成随机精确解向量
  std::vector<data_t> exact_solution = generateExactSolution(N);

  // 打印精确解的前10个元素
  std::cout << "Exact solution (first 10 elements):" << std::endl;
  for (int i = 0; i < std::min(10, N); ++i) {
    std::cout << std::fixed << std::setprecision(6) << exact_solution[i] << " ";
  }
  std::cout << std::endl << std::endl;

  // 转换为CSR格式并计算b = A * x_exact
  CUSPARSEMatrix A(N, N);
  A.convertFromDense(dense_matrix);
  std::vector<data_t> b(N);
  A.multiplyVector(exact_solution, b);
  A.moveToDevice();

  // 分配设备内存
  data_t *d_x, *d_b, *d_r, *d_norm;
  CHECK_CUDA(hipMalloc(&d_x, N * sizeof(data_t)));
  CHECK_CUDA(hipMalloc(&d_b, N * sizeof(data_t)));
  CHECK_CUDA(hipMalloc(&d_r, N * sizeof(data_t)));
  CHECK_CUDA(hipMalloc(&d_norm, sizeof(data_t)));

  // 初始化解向量和拷贝右端向量
  CHECK_CUDA(hipMemset(d_x, 0, N * sizeof(data_t)));
  CHECK_CUDA(
      hipMemcpy(d_b, b.data(), N * sizeof(data_t), hipMemcpyHostToDevice));

  auto start = std::chrono::high_resolution_clock::now();
  int iter = 0;
  data_t norm = 1.0f;
  const int threads = 256;
  const int blocks = (N + threads - 1) / threads;

  while (norm > EPS && iter < MAX_ITER) {
    // 执行一次Gauss-Seidel迭代
    gaussSeidelKernel<<<blocks, threads>>>(N, A.d_values, A.d_row_offsets,
                                           A.d_col_indices, d_b, d_x);

    CHECK_CUDA(hipDeviceSynchronize());

    // 计算残差
    computeResidual<<<blocks, threads>>>(N, A.d_values, A.d_row_offsets,
                                         A.d_col_indices, d_x, d_b, d_r);

    CHECK_CUDA(hipDeviceSynchronize());

    // 计算残差范数
    CHECK_CUDA(hipMemset(d_norm, 0, sizeof(data_t)));
    computeResidualNorm<<<blocks, threads, threads * sizeof(data_t)>>>(
        d_r, d_norm, N);

    CHECK_CUDA(
        hipMemcpy(&norm, d_norm, sizeof(data_t), hipMemcpyDeviceToHost));
    norm = std::sqrt(norm);

    std::cout << "Iteration " << iter << ", residual norm: " << norm
              << std::endl;
    iter++;
  }

  auto end = std::chrono::high_resolution_clock::now();
  auto duration =
      std::chrono::duration_cast<std::chrono::microseconds>(end - start);

  std::cout << "\nConverged in " << iter << " iterations, " << duration.count()
            << "us" << std::endl;
  std::cout << "Final residual norm: " << norm << std::endl;

  // 获取数值解
  std::vector<data_t> numerical_solution(N);
  CHECK_CUDA(hipMemcpy(numerical_solution.data(), d_x, N * sizeof(data_t),
                        hipMemcpyDeviceToHost));

  // 计算与精确解的平均误差
  data_t total_error = 0.0f;
  data_t max_error = 0.0f;
  for (int i = 0; i < N; ++i) {
    data_t error = std::abs(numerical_solution[i] - exact_solution[i]);
    total_error += error;
    max_error = std::max(max_error, error);
  }
  data_t avg_error = total_error / N;

  std::cout << "\nAverage error compared to exact solution: " << avg_error
            << std::endl;
  std::cout << "Maximum error compared to exact solution: " << max_error
            << std::endl;

  // 打印数值解的前10个元素
  std::cout << "\nNumerical solution (first 10 elements):" << std::endl;
  for (int i = 0; i < std::min(10, N); ++i) {
    std::cout << std::fixed << std::setprecision(6) << numerical_solution[i]
              << " ";
  }
  std::cout << std::endl;

  // 清理资源
  hipsparseDestroy(cusparse_handle);

  hipFree(d_x);
  hipFree(d_b);
  hipFree(d_r);
  hipFree(d_norm);

  return 0;
}