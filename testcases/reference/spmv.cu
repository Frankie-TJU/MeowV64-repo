// Author: Chu Xu
// Build with:
// /usr/local/cuda/bin/nvcc spmv.cu -o spmv -lcusparse -lnvidia-ml
// optionally, add -DN=xxx and -DNNZ=xxx

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>

#include <nvml.h>

#ifndef N
#define N 2048
#endif

#ifndef NNZ
#define NNZ 204800
#endif

__host__ __device__ static uint64_t lfsr63(uint64_t x) {
  uint64_t bit = (x ^ (x >> 1)) & 1;
  return (x >> 1) | (bit << 62);
}

#define CHECK_CUDA(call)                                                       \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      printf("CUDA error at %s:%d: %s\n", __FILE__, __LINE__,                  \
             hipGetErrorString(err));                                         \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

#define CHECK_CUSPARSE(call)                                                   \
  do {                                                                         \
    hipsparseStatus_t err = call;                                               \
    if (err != HIPSPARSE_STATUS_SUCCESS) {                                      \
      printf("cuSPARSE error at %s:%d: %s\n", __FILE__, __LINE__,              \
             hipsparseGetErrorString(err));                                     \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

#define CHECK_NVML(call)                                                       \
  do {                                                                         \
    nvmlReturn_t err = call;                                                   \
    if (err != NVML_SUCCESS) {                                                 \
      printf("NVML error at %s:%d: %s\n", __FILE__, __LINE__,                  \
             nvmlErrorString(err));                                            \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

int main() {
  // 创建输出文件
  FILE *fp = fopen("out_GPU.log", "w");
  if (fp == NULL) {
    printf("Error opening output file\n");
    return 1;
  }

  // NVML 初始化
  nvmlDevice_t device;
  CHECK_NVML(nvmlInit_v2());
  CHECK_NVML(nvmlDeviceGetHandleByIndex_v2(0, &device));

  // CPU内存分配和初始化
  float *h_val = new float[NNZ];
  uint32_t *h_idx = new uint32_t[NNZ];
  float *h_x = new float[N];
  uint32_t *h_ptr = new uint32_t[N + 1];
  float *h_y = new float[N];

  // 初始化数据（与原来相同的初始化过程）
  for (int i = 0; i < NNZ; i++) {
    h_val[i] = (float)i / (float)NNZ;
  }

  uint64_t seed = 1;
  for (int i = 0; i < NNZ; i++) {
    seed = lfsr63(seed);
    h_idx[i] = seed % N;
  }

  for (int i = 0; i < N; i++) {
    volatile float temp = (float)i / (float)N;
    h_x[i] = temp;
  }

  for (int i = 0; i < N; i++) {
    h_ptr[i] = i * (NNZ / N);
  }
  h_ptr[N] = NNZ;

  // GPU内存分配
  float *d_val, *d_x, *d_y;
  int *d_idx, *d_ptr;

  CHECK_CUDA(hipMalloc(&d_val, NNZ * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_idx, NNZ * sizeof(int)));
  CHECK_CUDA(hipMalloc(&d_x, N * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_ptr, (N + 1) * sizeof(int)));
  CHECK_CUDA(hipMalloc(&d_y, N * sizeof(float)));

  // 创建临时缓冲区进行类型转换
  int *h_idx_int = new int[NNZ];
  int *h_ptr_int = new int[N + 1];

  // 将uint32_t转换为int
  for (int i = 0; i < NNZ; i++) {
    h_idx_int[i] = static_cast<int>(h_idx[i]);
  }
  for (int i = 0; i <= N; i++) {
    h_ptr_int[i] = static_cast<int>(h_ptr[i]);
  }

  // 数据传输到GPU
  CHECK_CUDA(
      hipMemcpy(d_val, h_val, NNZ * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(
      hipMemcpy(d_idx, h_idx_int, NNZ * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_ptr, h_ptr_int, (N + 1) * sizeof(int),
                        hipMemcpyHostToDevice));

  // 创建cuSPARSE句柄
  hipsparseHandle_t handle;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  // 创建矩阵描述符（新版API）
  hipsparseSpMatDescr_t matA;
  hipsparseDnVecDescr_t vecX, vecY;
  void *dBuffer = NULL;
  size_t bufferSize = 0;
  float alpha = 1.0f;
  float beta = 0.0f;

  // 创建稀疏矩阵描述符
  CHECK_CUSPARSE(hipsparseCreateCsr(&matA, N, N, NNZ, d_ptr, d_idx, d_val,
                                   HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

  // 创建密集向量描述符
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, N, d_x, HIP_R_32F));
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, N, d_y, HIP_R_32F));

  // 创建CUDA计时器
  hipEvent_t start, stop;
  CHECK_CUDA(hipEventCreate(&start));
  CHECK_CUDA(hipEventCreate(&stop));

  // 获取缓冲区大小
  CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
      handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY,
      HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));

  // 分配缓冲区
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

  // 开始计时
  CHECK_CUDA(hipEventRecord(start));

  // 执行SpMV计算
  CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,
                              matA, vecX, &beta, vecY, HIP_R_32F,
                              HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));

  // 停止计时
  CHECK_CUDA(hipEventRecord(stop));
  CHECK_CUDA(hipEventSynchronize(stop));

  unsigned int power;
  CHECK_NVML(nvmlDeviceGetPowerUsage(device, &power));

  // 计算执行时间
  float milliseconds = 0;
  CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
  double seconds = milliseconds / 1000.0;

  // 将结果从GPU复制回CPU
  CHECK_CUDA(hipMemcpy(h_y, d_y, N * sizeof(float), hipMemcpyDeviceToHost));

  // 打印执行时间
  printf("Duration: %.10f s.\n", seconds);
  printf("Power: %.3f W.\n", (double)power / 1000);

  // 将结果写入文件
  fprintf(fp, "Result: [%f", h_y[0]);
  for (int i = 1; i < N; i++) {
    fprintf(fp, ", %f", h_y[i]);
  }
  fprintf(fp, "]\n");

  // 清理资源
  CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
  CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX));
  CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY));
  CHECK_CUSPARSE(hipsparseDestroy(handle));

  if (dBuffer) {
    CHECK_CUDA(hipFree(dBuffer));
  }

  fclose(fp);
  delete[] h_val;
  delete[] h_idx;
  delete[] h_x;
  delete[] h_ptr;
  delete[] h_y;
  delete[] h_idx_int;
  delete[] h_ptr_int;

  CHECK_CUDA(hipFree(d_val));
  CHECK_CUDA(hipFree(d_idx));
  CHECK_CUDA(hipFree(d_x));
  CHECK_CUDA(hipFree(d_ptr));
  CHECK_CUDA(hipFree(d_y));

  CHECK_CUDA(hipEventDestroy(start));
  CHECK_CUDA(hipEventDestroy(stop));

  return 0;
}
